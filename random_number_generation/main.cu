//
// Created by andreas on 31.07.22.
//

#include <hiprand.h>
#include "./../helper/memory_tracker.cuh"
int main()
{
	hiprandGenerator_t generator;
	unsigned long seed;
	hiprandStatus_t curand_result;
	hipError_t cuda_result;
	MemoryTracker<float> tracker;
	auto host_sample = tracker.allocate_host_memory(100, "host_sample");
	auto device_sample = tracker.allocate_device_memory(100, "device_sample");

	return 0;
}
