#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include "helper/memory_tracking.h"
#include "algorithms/vector_addition.cuh"


int main()
{
	// Error code to check return values for CUDA calls
	hipError_t error = hipSuccess;

	// Print the vector length to be used, and compute its size
	size_t number_of_elements = 50000;
	std::cout << "Vector addition of "<< number_of_elements << " elements \n";

	// Allocate the host vectors
	auto tracker = MemoryTracking<int>();
	auto array_A = tracker.allocate_host_memory(number_of_elements);
	auto array_B = tracker.allocate_host_memory(number_of_elements);
	auto array_C = tracker.allocate_host_memory(number_of_elements);

	// Initialize the host input vectors
	for (int i = 0; i < number_of_elements; ++i)
	{
		array_A[i] = rand()/(int)RAND_MAX;
		array_B[i] = rand()/(int)RAND_MAX;
	}

	auto device_array_A = tracker.allocate_device_memory(number_of_elements);
	auto device_array_B = tracker.allocate_device_memory(number_of_elements);
	auto device_array_C = tracker.allocate_device_memory(number_of_elements);


	printf("Copy input data from the host memory to the CUDA device\n");
	tracker.copy_host_array_to_device_array(array_A, device_array_A, number_of_elements);
	tracker.copy_host_array_to_device_array(array_B, device_array_B, number_of_elements);
	tracker.copy_host_array_to_device_array(array_C, device_array_C, number_of_elements);


	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(number_of_elements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	vectorAdd<int><<<blocksPerGrid, threadsPerBlock>>>(device_array_A, device_array_B, device_array_C, number_of_elements);
	error = hipGetLastError();

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	tracker.copy_device_array_to_host_array(array_C, device_array_C, number_of_elements);

	// Verify that the result vector is correct
	for (int i = 0; i < number_of_elements; ++i)
	{
		if (fabs(array_A[i] + array_B[i] - array_C[i]) > 1e-5 || array_A[i] == 0)
		{
			std::cout << "ERROR in result." << std::endl;
			exit(EXIT_FAILURE);
		}
	}
	std::cout << "Test PASSED " <<std::endl;


	tracker.free_device_memory(device_array_A);
	tracker.free_device_memory(device_array_B);
	tracker.free_device_memory(device_array_C);

	tracker.free_host_memory(array_A);
	tracker.free_host_memory(array_B);
	tracker.free_host_memory(array_C);

	// Free host memory
	return 0;
}
