#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "helper/memory_tracking.h"

template <typename T>
__global__ void
vectorAdd(const T *A, const T *B, T *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}

template <typename T>
void copy_host_array_to_device_array(T * host_memory, T* device_memory, size_t number_of_elements)
{
	hipError_t error = hipSuccess;
	auto size = number_of_elements*sizeof(T);
	error = hipMemcpy(device_memory, host_memory, size, hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to copy from host to device (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

template <typename T>
void copy_device_array_to_host_array(T * host_memory, T* device_memory, size_t number_of_elements)
{
	hipError_t error = hipSuccess;
	auto size = number_of_elements*sizeof(T);
	error = hipMemcpy(host_memory, device_memory, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to copy from device to host (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

}




int main()
{
	// Error code to check return values for CUDA calls
	hipError_t error = hipSuccess;

	// Print the vector length to be used, and compute its size
	size_t number_of_elements = 50000;
	std::cout << "Vector addition of "<< number_of_elements << " elements \n";

	// Allocate the host vectors
	auto tracker = MemoryTracking<float>();
	auto array_A = tracker.allocate_host_memory(number_of_elements);
	auto array_B = tracker.allocate_host_memory(number_of_elements);
	auto array_C = tracker.allocate_host_memory(number_of_elements);

	// Initialize the host input vectors
	for (int i = 0; i < number_of_elements; ++i)
	{
		array_A[i] = rand()/(float)RAND_MAX;
		array_B[i] = rand()/(float)RAND_MAX;
	}

	auto device_array_A = tracker.allocate_device_memory(number_of_elements);
	auto device_array_B = tracker.allocate_device_memory(number_of_elements);
	auto device_array_C = tracker.allocate_device_memory(number_of_elements);


	printf("Copy input data from the host memory to the CUDA device\n");
	copy_host_array_to_device_array<float>(array_A, device_array_A, number_of_elements);
	copy_host_array_to_device_array<float>(array_B, device_array_B, number_of_elements);
	copy_host_array_to_device_array<float>(array_C, device_array_C, number_of_elements);


	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(number_of_elements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	vectorAdd<float><<<blocksPerGrid, threadsPerBlock>>>(device_array_A, device_array_B, device_array_C, number_of_elements);
	error = hipGetLastError();

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	copy_device_array_to_host_array<float>(array_C, device_array_C, number_of_elements);

	// Verify that the result vector is correct
	for (int i = 0; i < number_of_elements; ++i)
	{
		if (fabs(array_A[i] + array_B[i] - array_C[i]) > 1e-5)
		{
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}
	std::cout << "Test PASSED " <<std::endl;


	tracker.free_device_memory(device_array_A);
	tracker.free_device_memory(device_array_B);
	tracker.free_device_memory(device_array_C);

	tracker.free_host_memory(array_A);
	tracker.free_host_memory(array_B);
	tracker.free_host_memory(array_C);

	// Free host memory
	return 0;
}
