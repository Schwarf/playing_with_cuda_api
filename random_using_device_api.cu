#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <cstdio>
#include <ctime>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <iostream>
#include "./helper/macros.h"
#include "./helper/memory_tracker.cuh"

#define N 1000000  // number of trials
#define DICE 20      // number of the maximum desired value

class Histogram
{
public:
	Histogram(size_t number_of_bins)
		:
		number_of_bins_(number_of_bins)
	{
		result_ = std::vector<long long unsigned>(number_of_bins, 0);
	}
	void add_values(unsigned int *input, size_t size)
	{
		for(size_t i =0; i < size; ++i)
		{
			result_[input[i]]++;
		}
	}
	void print_result(){
		for (size_t i = 0; i < DICE + 1; i++)
			printf("%2d : %10d\n", i, result_[i]);
	}
private:
	size_t number_of_bins_;
	std::vector<long long unsigned> result_;
};

__global__ void init(unsigned int seed, hiprandState_t *states)
{
	auto id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, /* the seed can be the same for each thread, here we pass the time from CPU */
				id,   /* the sequence number should be different for each core */
				0,    /* the offset is how much extra we advance in the sequence for each call, can be 0 */
				&states[id]);
}

__global__ void random_generation_with_ceiling(hiprandState_t *states, unsigned int *numbers)
{
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	numbers[id] = ceilf(hiprand_uniform(&states[id]) * DICE);

}


int main()
{
	auto histogram = Histogram(DICE);

	auto state_tracker = MemoryTracker<hiprandState_t>();
	hiprandState_t *states = state_tracker.allocate_device_memory(N, "device_curand_states");
	//hipMalloc((void **)&states, N * sizeof(hiprandState_t));


	// initialize the random states
	dim3 block_dimension = 1000;
	dim3 grdDim = (N + block_dimension.x - 1) / block_dimension.x;
	auto seed = time(0);
	std::cout << "Seed: " << seed <<std::endl;
	init<<<grdDim, block_dimension >>>(seed, states);

	// allocate an array of unsigned ints on the CPU and GPU
	auto random_number_tracker = MemoryTracker<unsigned int>();
	auto host_random_nums = random_number_tracker.allocate_host_memory(N, "host_random_numbers");
	auto device_random_nums = random_number_tracker.allocate_device_memory(N, "device_random_numbers");


	// get random number with ceiling
	random_generation_with_ceiling<<<grdDim, block_dimension >>>(states, device_random_nums);
	random_number_tracker.copy_device_array_to_host_array(host_random_nums, device_random_nums, N);

	printf("Histogram for random numbers generated with ceiling\n");
	histogram.add_values(host_random_nums, N);
	histogram.print_result();
	state_tracker.free_device_memory(states);
	//hipFree(device_random_nums);
	random_number_tracker.free_device_memory(device_random_nums);


	return 0;
}
